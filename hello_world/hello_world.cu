
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void print_from_gpu(void){
    printf("hellow world from GPU. From thread [%d,%d] \
		From device\n", threadIdx.x,blockIdx.x);
}

int main(void){
    printf("Hellow world from CPU\n");
    print_from_gpu<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}