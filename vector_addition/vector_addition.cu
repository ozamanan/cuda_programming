
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 10

void host_add(int *a, int *b, int *c){
    for(int i = 0; i < N; i++){
        c[i] = a[i] + b[i];
    }
}

void init_array(int *data){
    for(int i = 0; i < N; i++){
        data[i] = i*2;
    }
}

void print_result(int *a, int *b, int *c){
    for(int i = 0; i < N; i++)
    printf("%d + %d = %d \n", a[i], b[i], c[i]);
}

int main(void){
    int *a, *b, *c;
    int size = N*sizeof(int);

    //allocate space for host copies of a, b, c, and init the vectors
    a = (int *)malloc(size);
    init_array(a);
    b = (int *)malloc(size);
    init_array(b);
    c = (int *)malloc(size);
    
    //addition
    host_add(a, b, c);

    print_result(a, b, c);

    free(a); free(b); free(c);

    return 0;
}