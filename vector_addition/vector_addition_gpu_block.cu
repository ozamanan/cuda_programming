
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 10

void host_add(int *a, int *b, int *c){
    for(int i = 0; i < N; i++){
        c[i] = a[i] + b[i];
    }
}

__global__ void device_add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void init_array(int *data){
    for(int i = 0; i < N; i++){
        data[i] = i*2;
    }
}

void print_result(int *a, int *b, int *c){
    for(int i = 0; i < N; i++)
    printf("%d + %d = %d \n", a[i], b[i], c[i]);
}

int main(void){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    int size = N * sizeof(int);

    //allocate space for host copies of a, b, c and init the vectors
    a = (int *)malloc(size);
    init_array(a);
    b = (int *)malloc(size);
    init_array(b);
    c = (int *)malloc(size);

    //allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    device_add<<<N,1>>>(d_a, d_b, d_c);
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	print_result(a,b,c);

	free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}